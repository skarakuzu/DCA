#include "hip/hip_runtime.h"
// Copyright (C) 2018 ETH Zurich
// Copyright (C) 2018 UT-Battelle, LLC
// All rights reserved.
//
// See LICENSE.txt for terms of usage.
// See CITATION.txt for citation guidelines if you use this code for scientific publications.
//
// Author: Giovanni Balduzzi (gbalduzz@itp.phys.ethz.ch)
//
// This file implements G4Helper::set.

#include "dca/phys/dca_step/cluster_solver/ctaux/accumulator/tp/TpEqTime_helper.cuh"

#include <algorithm>
#include <array>
#include <mutex>
#include <stdexcept>

namespace dca {
namespace phys {
namespace solver {
namespace ctaux {
// dca::phys::solver::ctaux::

__device__ __constant__ TpEqTimeHelper tpeqtime_helper;

void TpEqTimeHelper::set(const int* sub_r, int lds, int nr_sub, const int* G0_indices_up, int ldG0_indices_up, const int* G0_indices_dn, int ldG0_indices_dn, const float* G0_sign_up, int ldG0_sign_up,  const float* G0_sign_dn, int ldG0_sign_dn, const float* G0_integration_factor_up, int ldG0_integration_factor_up, const float* G0_integration_factor_dn, int ldG0_integration_factor_dn, const float* G0_original_up, int ldG0_original_up, const float* G0_original_dn, int ldG0_original_dn,  int G0dmnsize, int tVertex_dmnsize,  const double* akima_coeff, int lakm, int nb_akm, int ns_akm, int nr_akm, int nt_akm, int akima_size, int* fixed_config_b_ind, int* fixed_config_r_ind, int* fixed_config_t_ind, double* fixed_config_t_val, double beta, double N_div_beta) {


  static std::once_flag flag;

  std::call_once(flag, [=]() {
    TpEqTimeHelper host_helper;
    host_helper.bVrtxdmn_ = G0dmnsize;
    host_helper.tVrtxdmn_ = tVertex_dmnsize;
    host_helper.rDmnt_ = nr_akm;
    host_helper.lds_ = lds;
    host_helper.ldG0_indices_up_ = ldG0_indices_up;
    host_helper.ldG0_indices_dn_ = ldG0_indices_dn;
    host_helper.ldG0_sign_up_ = ldG0_sign_up;
    host_helper.ldG0_sign_dn_ = ldG0_sign_dn;
    host_helper.ldG0_integration_factor_up_ = ldG0_integration_factor_up;
    host_helper.ldG0_integration_factor_dn_ = ldG0_integration_factor_dn;
    host_helper.ldG0_original_up_ = ldG0_original_up;
    host_helper.ldG0_original_dn_ = ldG0_original_dn;
    host_helper.N_div_beta_ = N_div_beta;
    host_helper.beta_ = beta;

/*
    host_helper.ext_size_ = 0;
    for (const int idx : delta_w)
      host_helper.ext_size_ = std::max(host_helper.ext_size_, std::abs(idx));
*/

    const std::array<int, 7> akima_sizes{lakm,
				    nb_akm,
                                    ns_akm,
                                    nb_akm,
                                    ns_akm,
                                    nr_akm,
                                    nt_akm};

    std::array<int, 7> steps;
    steps[0] = 1;
    for (std::size_t i = 1; i < steps.size(); ++i)
      steps[i] = steps[i - 1] * akima_sizes[i - 1];

    std::copy_n(steps.data(), steps.size(), host_helper.akima_steps_);



    const std::array<int, 4> chi_sizes{nb_akm,
				  nb_akm,
				  nr_akm,	
				  tVertex_dmnsize};

    std::array<int, 4> steps2;
    steps2[0] = 1;
    for (std::size_t i = 1; i < steps2.size(); ++i)
      steps2[i] = steps2[i - 1] * chi_sizes[i - 1];


    std::copy_n(steps2.data(), steps2.size(), host_helper.chi_steps_);


    hipMalloc(&host_helper.sub_matrix_, sizeof(int) * lds * nr_sub);
    hipMemcpy(host_helper.sub_matrix_, sub_r, sizeof(int) * lds * nr_sub, hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_indices_up_, sizeof(int) * ldG0_indices_up * G0dmnsize);
    hipMemcpy(host_helper.G0_indices_up_, G0_indices_up, sizeof(int) * ldG0_indices_up * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_indices_dn_, sizeof(int) * ldG0_indices_dn * G0dmnsize);
    hipMemcpy(host_helper.G0_indices_dn_, G0_indices_dn, sizeof(int) * ldG0_indices_dn * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_sign_up_, sizeof(float) * ldG0_sign_up * G0dmnsize);
    hipMemcpy(host_helper.G0_sign_up_, G0_sign_up, sizeof(float) * ldG0_sign_up * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_sign_dn_, sizeof(float) * ldG0_sign_dn * G0dmnsize);
    hipMemcpy(host_helper.G0_sign_dn_, G0_sign_dn, sizeof(float) * ldG0_sign_dn * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_integration_factor_up_, sizeof(float) * ldG0_integration_factor_up * G0dmnsize);
    hipMemcpy(host_helper.G0_integration_factor_up_, G0_integration_factor_up, sizeof(float) * ldG0_integration_factor_up * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_integration_factor_dn_, sizeof(float) * ldG0_integration_factor_dn * G0dmnsize);
    hipMemcpy(host_helper.G0_integration_factor_dn_, G0_integration_factor_dn, sizeof(float) * ldG0_integration_factor_dn * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_original_up_, sizeof(float) * ldG0_original_up * G0dmnsize);
    hipMemcpy(host_helper.G0_original_up_, G0_original_up, sizeof(float) * ldG0_original_up *G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_original_dn_, sizeof(float) * ldG0_original_dn *G0dmnsize);
    hipMemcpy(host_helper.G0_original_dn_, G0_original_dn, sizeof(float) * ldG0_original_dn *G0dmnsize,
               hipMemcpyHostToDevice);
/*
    hipMalloc(&host_helper.G0_indices_up_, sizeof(int) * G0dmnsize * G0dmnsize);
    hipMemcpy(host_helper.G0_indices_up_, G0_indices_up, sizeof(int) * G0dmnsize * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_indices_dn_, sizeof(int) * G0dmnsize * G0dmnsize);
    hipMemcpy(host_helper.G0_indices_dn_, G0_indices_dn, sizeof(int) * G0dmnsize * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_sign_up_, sizeof(float) * G0dmnsize * G0dmnsize);
    hipMemcpy(host_helper.G0_sign_up_, G0_sign_up, sizeof(float) * G0dmnsize * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_sign_dn_, sizeof(float) * G0dmnsize * G0dmnsize);
    hipMemcpy(host_helper.G0_sign_dn_, G0_sign_dn, sizeof(float) * G0dmnsize * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_integration_factor_up_, sizeof(float) * G0dmnsize * G0dmnsize);
    hipMemcpy(host_helper.G0_integration_factor_up_, G0_integration_factor_up, sizeof(float) * G0dmnsize * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_integration_factor_dn_, sizeof(float) * G0dmnsize * G0dmnsize);
    hipMemcpy(host_helper.G0_integration_factor_dn_, G0_integration_factor_dn, sizeof(float) * G0dmnsize * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_original_up_, sizeof(float) * G0dmnsize * G0dmnsize);
    hipMemcpy(host_helper.G0_original_up_, G0_original_up, sizeof(float) * G0dmnsize *G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.G0_original_dn_, sizeof(float) * G0dmnsize *G0dmnsize);
    hipMemcpy(host_helper.G0_original_dn_, G0_original_dn, sizeof(float) * G0dmnsize *G0dmnsize,
               hipMemcpyHostToDevice);

*/


    hipMalloc(&host_helper.akima_coefficients_, sizeof(double) * akima_size);
    hipMemcpy(host_helper.akima_coefficients_, akima_coeff, sizeof(double) * akima_size,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.fixed_config_b_ind_, sizeof(int) * G0dmnsize);
    hipMemcpy(host_helper.fixed_config_b_ind_, fixed_config_b_ind, sizeof(int) * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.fixed_config_r_ind_, sizeof(int) * G0dmnsize);
    hipMemcpy(host_helper.fixed_config_r_ind_, fixed_config_r_ind, sizeof(int) * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.fixed_config_t_ind_, sizeof(int) *G0dmnsize);
    hipMemcpy(host_helper.fixed_config_t_ind_, fixed_config_t_ind, sizeof(int) * G0dmnsize,
               hipMemcpyHostToDevice);

    hipMalloc(&host_helper.fixed_config_t_val_, sizeof(double) * G0dmnsize);
    hipMemcpy(host_helper.fixed_config_t_val_, fixed_config_t_val, sizeof(double) * G0dmnsize,
               hipMemcpyHostToDevice);


    hipMemcpyToSymbol(HIP_SYMBOL(tpeqtime_helper), &host_helper, sizeof(TpEqTimeHelper));
  });
}

}  // namespace ctaux
}  // namespace solver
}  // namespace phys
}  // namespace dca
